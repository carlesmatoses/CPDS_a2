#include <math.h>
#include <float.h>
#include <hip/hip_runtime.h>

__global__ void gpu_Heat (float *h, float *g, int N) {

	// TODO: kernel computation
	int np = N;
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;

	if (i > 0 && i < np-1 && j > 0 && j < np-1) {
		g[i*np+j] = 0.25 * (h[(i-1)*np+j] + h[(i+1)*np+j] + h[i*np+(j-1)] + h[i*np+(j+1)]);
	}
}
